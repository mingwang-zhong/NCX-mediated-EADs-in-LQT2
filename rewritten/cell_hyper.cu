#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
using namespace std;

/////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////
// #define Permeabilized		// Permeabalized cell. No sarcolemmal ion channels
// #define PermeabilizedB		// Permeabalized cell. Myocyte diffuses with a bath ( boundary condition )
#define LQT2			// Long-QT 2 syndrome simulation. No I_Kr
#define ISO			// Isoproterenol, increases Uptake and I_Ca,L (and/or IKs)
// #define Vclamp		// step function voltage clamp
// #define APclamp		// action potential clamp

/////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////
#define DT 	0.025 //ms, time step
#define stoptime (60001.0) //ms
#define PCL 4000.0	//ms, pacing cycle length
#define stopbeat 15
#define time_before_beat 100 //ms

#define out_step	100 // number of steps to output data
// #define output_linescan // output linescan
//////////////////////////////////////////////////////////////////
////////////////// CUDA block size 

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 7
#define BLOCK_SIZE_Z 4
#define Nx 	64	// Number of CRUs in the x direction
#define Ny 	28	// Number of CRUs in the y direction
#define Nz 	12	// Number of CRUs in the z direction
#define Nix	2	// number of lattices in x direction in a CRU
#define Niy	2	// number of lattices in y direction in a CRU
#define Niz	2	// number of lattices in z direction in a CRU
#define Nci	8 //(Nix*Niy*Niz) // number of lattices in a CRU
#define DX	(1.8/Nix) // um, size of each lattice in longitudinal direction
#define DY	(0.9/Niy) // um
#define DZ	(0.9/Niz) // um

//////////////////////////////////////////////////////////////////
////////////////// cell properties

#define Vp 0.00126 	//um^3, Volume of the proximal space
#define Vs 0.025 //um^3, Volume of the submembrane space
#define Vjsr 0.02	//um^3, Volume of the JSR space
#define Vi (0.5/Nci)	//um^3, Volume of the cytosolic space, for each compartment
#define Vnsr (0.025/Nci)	//um^3, Volume of the NSR space
#define taups 0.0283	//ms, Diffusion time from the proximal to the submembrane
#define taupi 0.1 //ms, Diffusion time from the proximal to the cytosol
#define tausi 0.04		//ms, Diffusion time from the submembrane to the cytosolic
#define taust 1.42  //ms, diffusion time in submembrane along transverse direction
#define tautr 6.25	//ms, Diffusion time from NSR to JSR 
#define taunl 4.2		//ms, diffusion time of longitudinal NSR
#define taunt 1.26		//ms, diffusion time of transverse NSR
#define tauil 0.98	//ms, diffusion time of longitudinal cytosolic
#define tauit 0.462 	//ms, diffusion time of transverse cytosolic
#define Ddye 	2.0	// diffusion time prefactor of dye

#ifdef Permeabilized
	#define ci_basal (atof(argv[2]))
	#define cjsr_basal (atof(argv[3]))
#else
	#define ci_basal 0.0944
	#define cjsr_basal 400.0
#endif


/////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////
//////////// ion channel parameters 

#define	gKs  (0.2)  // mS/uF, IKs conductance
#define gtof (0.1)  // mS/uF, Itof conductance
#define gtos (0.04)  // mS/uF, Itos conductance
#define gNaK (1.5)  // mS/uF, INaK conductance
#define gK1  (0.6)  // mS/uF, IK1 conductance
#define gNa  (12.0)  // mS/uF, INa conductance
#define gNaLeak (0.0015)  // mS/uF, INaLeak conductance
#define alphaNaL 0
#define Vncx 	(21.0*2.5) //uM/ms, strength of NCX current
#define	Vleak (0.00212/2) // ms^{-1}, Shannon et al 2004, Eq. 107 (0.00212 = 5.348e-6*0.5/0.00126 )

#ifdef ISO
	#define Vup (0.3*1.75) // uM/ms, uptake strength
#else
	#define Vup (0.3) // uM/ms
#endif

#ifdef LQT2
	#define	gKr 0
#else
	#define	gKr 0.01  // mS/uF, IKs conductance
#endif

// ryr gating
#define nryr 	100		//Number of Ryr channels
#define f_Jmax 	11.5	//Jmax prefactor
#define tauu 2000.0  // ms, transition rate from CSQN-bound states to CSQN-unbound states
#define taub 2.0 	    // ms, transition rate from CSQN-unbound states to CSQN-bound states
#define taucu 1.0    // ms, transition rate from open-unbound state to closed-unbound state
#define taucb 1.0    // ms, transition rate from open-bound state to closed-bound state
#define Spark_Threshold 3000.0 // uM/ms, when RyR release flux is larger than this, it is a spark.
#define Sparks_Interval 100.0 // ms, minimum time interval between sparks

// luminal gating
#define nCa 22.0 // number of Ca2+ binding sites of each CSQN molecule
#define BCSQN	460.0 //uM, concentration of CSQN
#define Kc 600.0 //uM, Dissociation constant of CSQN

// LCC ica
#define	f_ica (0.64)  // prefactor of single LCC current
#define Pca (17.85*f_ica)	// umol/C/ms, 11.9: Restrepo 2008
#define NLCC 4	 // number of LCC channels in each dyadic space
#define gammai 0.341 // Activity coefficient of Ca2+

// NCX
#define	NaO 	140.0 // mM, [Na+]o
#define Kmcai 	0.00359 // uM
#define Kmcao 	1.3 // mM
#define Kmnai 	12.3 // mM
#define Kmnao 	87.5 // mM
#define eta		0.35
#define ksat	0.27

// other
#define CaO 1.8		//mM, external Ca2+ concentration
#define KI 	140.0	//mM, internal K+ concentration
#define KO 	5.40	//mM, external K+ concentration
#define Ek  ( (1.00/FRT)*log(KO/KI) )	//mV
#define Cm 	45 // pF, Capacitance of the whole cell membrane
#define Faraday 96.485		//	C/mmol
#define RR	8.314			//	J/mol/K
#define Temperature	308		//	K
#define FRT (Faraday/RR/Temperature)
#define PI 	3.1415926

#define pos(x,y,z)		(Nx*Ny*(z)+Nx*(y)+(x))
#define posi(i,j,k)		(Nix*Niy*(k)+Nix*(j)+(i))	// position in a CRU
#define posall(i,j,k)	((k)*(Nx*Nix)*(Ny*Niy)+(j)*(Nx*Nix)+(i))	// whole cell position
#define posallf(i,j,k)	((k)*(Nx*Nix/2+1)*Ny*Niy+(j)*(Nx*Nix/2+1)+(i))// whole cell position in k space(FFT)

#define pow2(x) ((x)*(x))
#define pow3(x) ((x)*(x)*(x))
#define pow4(x) ((x)*(x)*(x)*(x))
//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////

#define ktson 	0.00254 // (uM*ms)^(-1), associate constant
#define ktsoff 	0.000033 // ms^(-1), dissociate constant
#define Bts 	134.0	//uM, Mg binding/unbinding does not occur very much. Bts = steady state free+cabound troponin.
#define ktfon 	0.0327
#define ktfoff 	0.0196 // Troponin fast
#define Btf 	70.0

#define kcalon 	0.0543 // Calmodulin
#define kcaloff 0.238
#define Bcal 	24.0

#define ksron 	0.1 // SR
#define ksroff 	0.06
#define Bsr 	19.0

#define ksaron 	0.1 // Sarcolemma
#define ksaroff 1.3
#define Bsar 	(42*(Vi*Nci/Vs)*1.2) // in Bers book, it is 42 uM/l cytosol

#define ksarhon 	0.1  // Membrane/High
#define ksarhoff 	0.03
#define Bsarh 		(15.0*(Vi*Nci/Vs)*1.2)

#define Bmyo 	140.0 // Myosin
#define konmyomg 	0.0000157
#define koffmyomg 	0.000057
#define konmyoca 	0.0138
#define koffmyoca 	0.00046
#define Mgi 	500.0 // Mg
#define Kmyomg 	(koffmyomg/konmyomg)
#define Kmyoca 	(koffmyoca/konmyoca)

#define kdyeon		0.08 // Dye
#define kdyeoff		0.09
#define Bdye		0//40.0

struct sl_bu{
	double casar; // uM, Ca bound Sarcolemma buffer concentration in submembrane space
	double casarh; // uM, Ca bound Membrane/High
	double cadye; // uM, Ca bound Dye
	double cadyenext; // uM, Ca bound Dye

	double casarj; // uM, Ca bound Sarcolemma in dyad
	double casarhj; // uM, Ca bound Membrane/High in dyad
	double cadyej; // uM, Ca bound Dye in dyad
	double cadyejnext; // uM, Ca bound Dye in dyad
};

struct cyt_bu{ // cytosolic buffers
	double cacal;
	double catf;
	double cats;
	double casr;
	double camyo;
	double mgmyo;
	double cadye;
	double cadyenext;
};

struct cytosol{
	double Juptake; // uM/ms, SERCA uptake flux
	double Jxileak;
	double ci; // uM
	double cinext;
	double cnsr;
	double cnsrnext;
};

struct cru{
	double JNCX; // uM/ms, NCX flux
	double JCa; // uM/ms, ICa flux
	double Jbg; // uM/ms, sarcolemma background current flux
	double cs;
	double csnext;
};

struct cru2{
	double cp;
	double cpnext;
	double cjsr;
	double Tcj; // uM, total Ca2+ in jSR

	int lcc[NLCC];
	int nLCC_open;

	double Jrel; // uM/ms, SR release flux via RyRs
	double Jleak; // uM/ms, leak flux from JSR to dyad
	int nou;
	int ncu;
	int nob;
	int ncb;

	double Ka;	// for Inaca

	hiprandState state;
};

__global__ void	setup_kernel(unsigned long long seed,cru2 *CRU2);
__global__ void Initial( cru *CRU, cru2 *CRU2, cytosol *CYT, cyt_bu *CBU, sl_bu *SBU, double ci_b, double cj_b);
__global__ void Compute( cru *CRU, cru2 *CRU2, cytosol *CYT, cyt_bu *CBU, sl_bu *SBU, double v, int step, double nai );
__global__ void Finish( cru *CRU, cru2 *CRU2, cytosol *CYT, cyt_bu *CBU, sl_bu *SBU );

__device__ int ryrgating(double cp, double cjsr, hiprandState *state, int *ncu, int *nou, int *ncb, int *nob, int i, int j, int k, int step);
__device__ int number_RyR_transit(hiprandState *state, int NN, double probability, int upBound);
__device__ int LCCgating(double v, double cp, hiprandState *state, int i );
__device__ double Single_LCC_Current(double v, double cp); // cp in mM
__device__ double ncx(double v, double cs, double nai, double *Ka);
__device__ double uptake(double ci, double cnsr);

double Ina( double v, double *hh, double *jj, double *mm, double nai );
double Ikr( double v, double *Xkr );
double Iks( double v, double *Xs1, double *Xs2, double *Qks, double cst, double nai );
double Ik1( double v );
double Itos(double v, double *Xtos, double *Ytos);
double Itof(double v, double *Xtof, double *Ytof);
double Inak( double v, double nai );
double sodium(double v, double nai, double I_Na, double I_NaK, double I_NCX);

void matrix2file(cytosol *CYT, int step);

int main(int argc, char **argv)
{
	int CudaDevice = 0;	
	if( argc >= 1 ) 
		CudaDevice = atoi(argv[1]);
	hipSetDevice(CudaDevice);

	size_t ArraySize_cru = Nx*Ny*Nz*sizeof(cru);		// CRU
	size_t ArraySize_cru2= Nx*Ny*Nz*sizeof(cru2);
	size_t ArraySize_cyt = Nci*Nx*Ny*Nz*sizeof(cytosol);	// cytosol space, Nci=Nci
	size_t ArraySize_cbu = Nci*Nx*Ny*Nz*sizeof(cyt_bu);	// cytosol space for the buffers
	size_t ArraySize_sbu = Nx*Ny*Nz*sizeof(sl_bu);		// submembrane space for buffers
	size_t ArraySize_dos = Nx*Ny*Nz*sizeof(double);		// total size of submembrane lattices
	size_t ArraySize_dol = Nci*Nx*Ny*Nz*sizeof(double);	// total # of cytosol lattices

	// Allocate arrays memory in CPU 
	cru *h_CRU;
	cru2 *h_CRU2;
	cytosol *h_CYT;
	cyt_bu *h_CBU;
	sl_bu *h_SBU;
	double *spark_clock;
	
	h_CRU = (cru*) malloc(ArraySize_cru);
	h_CRU2 = (cru2*) malloc(ArraySize_cru2);
	h_CYT = (cytosol*) malloc(ArraySize_cyt);
	h_CBU = (cyt_bu*) malloc(ArraySize_cbu);
	h_SBU = (sl_bu*) malloc(ArraySize_sbu);
	spark_clock = (double*) malloc(ArraySize_dos);
	
	//Allocate arrays in GPU
	cru *d_CRU;
	cru2 *d_CRU2;
	cytosol *d_CYT;
	cyt_bu *d_CBU;
	sl_bu *d_SBU;

	hipMalloc((void**)&d_CRU, ArraySize_cru);
	hipMalloc((void**)&d_CRU2,ArraySize_cru2);
	hipMalloc((void**)&d_CYT, ArraySize_cyt);
	hipMalloc((void**)&d_CBU, ArraySize_cbu);
	hipMalloc((void**)&d_SBU, ArraySize_sbu);

	/////////////////////////////////// variables /////////////////////////////////////////////////
	int step = 0;
	int i, j, k, ix; // i,j,k for CRU index; ix, jy, kz for lattices in each CRU
	double start_time = clock()/(1.0*CLOCKS_PER_SEC),    end_time;

	double nai = 6.0;
	double CaExt = 0, TotalCa = 0, TotalCa_before = 0;

	double v = -80.00;	// voltage
	double mm = 0.0010, hh = 1.00, jj = 1.00;	// INa 
	double Xkr = 0.0; // IKr
	double Xs1 = 0.08433669901, Xs2 = Xs1, Qks = 0.2;	// IKs 
	double Xtos = 0.01, Ytos = 1.0;	// Itos
	double Xtof = 0.02, Ytof = 0.8;	// Itof
	double I_NaK = 0, I_Na = 0, I_Kr = 0, I_Ks = 0, I_K1 = 0, I_tos = 0, I_tof = 0, I_Ca = 0, I_NCX = 0, I_bg = 0;

	double cit, cpt, cst, cjsrt, cnsrt ;
	int Nxyz = (Nx-2)*(Ny-2)*(Nz-2);
	
	/////// to calculate spark rate
	double num_spark = 0, spark_rate = 0;
	for (k=0;k<Nz;k++)
	{
		for (j=0;j<Ny;j++)
		{
			for (i=0;i<Nx;i++)
			{
				spark_clock[pos(i,j,k)] = Sparks_Interval;
			}
		}
	}

	///////////////////////////////////////////// files /////////////////////////////////////////////////

	FILE * wholecell_file = fopen("wholecell.txt","w");
	
	#ifdef output_linescan
		FILE * linescan_file = fopen("linescan.txt","w");
	#endif
	////////////////////////////////////////////////////////////////////////////////////////////////////	
	// Set paramaters for geometry of computation
	dim3 threadsPerBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 numBlocks(Nx/threadsPerBlock.x, Ny/threadsPerBlock.y, Nz/threadsPerBlock.z);

	setup_kernel<<<numBlocks, threadsPerBlock>>>(18,d_CRU2);
	Initial<<<numBlocks, threadsPerBlock>>>(d_CRU, d_CRU2, d_CYT, d_CBU, d_SBU, ci_basal, cjsr_basal);
	
	while ( step*DT < stoptime )
	{
		hipMemcpy(h_CRU, d_CRU, ArraySize_cru, hipMemcpyDeviceToHost);

		//////////////////////////////// whole cell average: cst, Ica, Inaca, Ibcg ///////////////////////
		
		if ( step%out_step==1 )
		{
			CaExt=0;	// total Ca2+ exchange through the cell membrane
		}

		cst = 0;
		I_Ca = 0;
		I_bg = 0;
		I_NCX = 0;
		
		for (k = 1; k < Nz-1; k++)
		{
			for (j = 1; j < Ny-1; j++)
			{
				for (i = 1; i < Nx-1; i++) 
				{
					cst += h_CRU[pos(i,j,k)].cs;
					I_Ca += h_CRU[pos(i,j,k)].JCa;
					I_bg += h_CRU[pos(i,j,k)].Jbg;
					I_NCX += h_CRU[pos(i,j,k)].JNCX;

					CaExt = CaExt - h_CRU[pos(i,j,k)].JCa*Vp*DT/Nxyz 
							+ h_CRU[pos(i,j,k)].JNCX*Vs*DT/Nxyz;
				}
			}
		}
		cst=cst/Nxyz;

		// Firstly, I convert uM/ms to pA, then I make it divided by capacitance (Cm, pF)
		// So the unit of I_Ca is pA/pF, equal to mV/ms, which is the unit of dv/dt
		I_Ca = I_Ca*0.0965*Vp*2.0 / Cm;
		I_NCX = I_NCX*0.0965*Vs / Cm;
		I_bg = I_bg*0.0965*Vp*2.0 / Cm;

		//////////////////// other ion channels ///////////////////
		I_Na = Ina(v, &hh, &jj, &mm, nai);
		I_Kr = Ikr(v, &Xkr);
		I_Ks = Iks(v, &Xs1, &Xs2, &Qks, cst, nai );
		I_K1 = Ik1(v);
		I_tos = Itos(v, &Xtos, &Ytos);
		I_tof = Itof(v, &Xtof, &Ytof);
		I_NaK = Inak(v, nai);
		nai = sodium(v, nai, I_Na, I_NaK, I_NCX);

		///////////////////////////	Action Potential ///////////////////////////////////
		double stim = 0;
		if( fmod(step*DT+PCL-time_before_beat,PCL) < 1.0 && step*DT > time_before_beat && step*DT < PCL*stopbeat )
			stim = 80.0;
		double dvh = -( I_Na + I_K1 + I_Kr + I_Ks + I_tos + I_tof + I_NCX + I_Ca + I_NaK + I_bg ) + stim; 
		v += dvh*DT;

		#ifdef Permeabilized
			v = -86;
		#endif
	
		#ifdef Vclamp
			v = -86;
			if( step*DT > time_before_beat && step*DT < time_before_beat+200 )
				v = 0;
		#endif

		#ifdef APclamp
			v = varray[((int)( t/DT+0.1 ))%((int)(PCL/DT+0.1))];
		#endif
		
		///////////////////////////////////////////////////////////////////////////////
		//////////////////////////////// output ///////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////
		if ( step%out_step==0 )
		{
			hipMemcpy(h_CRU2,d_CRU2,ArraySize_cru2,hipMemcpyDeviceToHost);
			hipMemcpy(h_CYT, d_CYT, ArraySize_cyt, hipMemcpyDeviceToHost);
			hipMemcpy(h_CBU, d_CBU, ArraySize_cbu, hipMemcpyDeviceToHost);
			hipMemcpy(h_SBU, d_SBU, ArraySize_sbu, hipMemcpyDeviceToHost);

		//	matrix2file(h_CYT, step);

			cit = 0;
			cpt = 0;
			cjsrt = 0;
			cnsrt = 0;
			TotalCa = 0;

			double catft=0, catst=0, casrt=0, camyot=0, mgmyot=0, cacalt=0, cadyet=0;
			double casart = 0, casarht = 0, cadyest = 0;
			double casarjt = 0, casarhjt = 0, cadyejt = 0;
			double Jleakt=0, Juptaket=0, Jrelt=0;
			int nout = 0, nobt = 0, ncut = 0, ncbt = 0;
			double icaflux = 0, ncxflux = 0, ncxfwd = 0, Kat = 0;
			double cjt=0;	// total Ca2+ in jSR
			double nlcc_open = 0, nlcc_flip = 0, nlcc_act = 0, nlcc_v = 0, nlcc_Ca = 0;

			int ps = 0;
			for (k = 1; k < Nz-1; k++)
			{
				for (j = 1; j < Ny-1; j++)
				{
					for (i = 1; i < Nx-1; i++) 
					{	
						ps=pos(i,j,k);

						if ( h_CRU[ps].JNCX < 0 )
							ncxfwd += h_CRU[ps].JNCX;

						icaflux += h_CRU[ps].JCa;
						ncxflux += h_CRU[ps].JNCX;
						Kat += h_CRU2[ps].Ka;
						cpt += h_CRU2[ps].cp;
						cjsrt += h_CRU2[ps].cjsr;
						cjt += h_CRU2[ps].Tcj;
						Jrelt += h_CRU2[ps].Jrel;
						Jleakt += h_CRU2[ps].Jleak;
						nout += h_CRU2[ps].nou;
						nobt += h_CRU2[ps].nob;
						ncut += h_CRU2[ps].ncu;
						ncbt += h_CRU2[ps].ncb;

						casart += h_SBU[ps].casar;
						casarht += h_SBU[ps].casarh;
						cadyest += h_SBU[ps].cadye;
						casarjt += h_SBU[ps].casarj;
						casarhjt += h_SBU[ps].casarhj;
						cadyejt += h_SBU[ps].cadyej;

						for ( ix = 0; ix < Nci; ++ix )
						{
							cit += h_CYT[ps*Nci+ix].ci/Nci;
							cnsrt += h_CYT[ps*Nci+ix].cnsr/Nci;
							catft += h_CBU[ps*Nci+ix].catf/Nci;
							catst += h_CBU[ps*Nci+ix].cats/Nci;
							casrt += h_CBU[ps*Nci+ix].casr/Nci;
							camyot += h_CBU[ps*Nci+ix].camyo/Nci;
							mgmyot += h_CBU[ps*Nci+ix].mgmyo/Nci;
							cacalt += h_CBU[ps*Nci+ix].cacal/Nci;
							cadyet += h_CBU[ps*Nci+ix].cadye/Nci;
							Juptaket += h_CYT[ps*Nci+ix].Juptake/Nci;
							if( h_CYT[ps*Nci+ix].ci > 50.0 )
							{
								cout << step*DT << " " << i << " " << j << " " << k << " " 
									 << ix << " error! ci=" << h_CYT[ps*Nci+ix].ci << endl;
							}
						}

						nlcc_open += h_CRU2[ps].nLCC_open;
						for( int ll = 0; ll < NLCC; ll++ )
						{
							switch ( h_CRU2[ps].lcc[ll] )
							{
								case 1: ++nlcc_flip; break;
								case 2: ++nlcc_act; break;
								case 3: ++nlcc_flip; ++nlcc_act; break;
								case 4: ++nlcc_v; break;
								case 5: ++nlcc_flip; ++nlcc_v; break;
								case 6: ++nlcc_act; ++nlcc_v; break;
								case 7: ++nlcc_flip; ++nlcc_act; ++nlcc_v; break;
								case 8: ++nlcc_Ca; break;
								case 9: ++nlcc_flip; ++nlcc_Ca; break;
								case 10: ++nlcc_act; ++nlcc_Ca; break;
								case 11: ++nlcc_flip; ++nlcc_act; ++nlcc_Ca; break;
								case 12: ++nlcc_v; ++nlcc_Ca; break;
								case 13: ++nlcc_flip; ++nlcc_v; ++nlcc_Ca; break;
								case 14: ++nlcc_act; ++nlcc_v; ++nlcc_Ca; break;
								case 15: ++nlcc_flip; ++nlcc_act; ++nlcc_v; ++nlcc_Ca; break;
							}
						}
						
						
					}
				}
			}
			
			cit /= Nxyz;
			cpt /= Nxyz;
			cjsrt /= Nxyz;
			cjt /= Nxyz;
			cnsrt /= Nxyz;
			catft /= Nxyz;
			catst /= Nxyz;
			casrt /= Nxyz;
			camyot /= Nxyz;
			mgmyot /= Nxyz;
			cacalt /= Nxyz;
			cadyet /= Nxyz;
			Jleakt /= Nxyz;
			Juptaket /= Nxyz;
			Jrelt /= Nxyz;
			ncxflux /= Nxyz;
			ncxfwd /= Nxyz;
			icaflux /= Nxyz;
			Kat /= Nxyz;
			casart /= Nxyz;
			casarht /= Nxyz;
			cadyest /= Nxyz;
			casarjt /= Nxyz;
			casarhjt /= Nxyz;
			cadyejt /= Nxyz;
			nlcc_open /= Nxyz;
			nlcc_flip /= Nxyz;
			nlcc_act /= Nxyz;
			nlcc_v /= Nxyz;
			nlcc_Ca /= Nxyz;
			
			TotalCa =	( cit+ catft + catst + casrt + camyot + cacalt + cadyet )*Vi*Nci 
						+ ( cst + casart + casarht + cadyest )*Vs 
						+ ( cpt + casarjt + casarhjt + cadyejt )*Vp 
						+ cjt*Vjsr
						+ cnsrt*Vnsr*Nci;

			//////////////////////////////////// spark rate /////////////////////////////////
			num_spark = 0.0;
			for (k=0;k<Nz;k++)
			{
				for (j=0;j<Ny;j++)
				{
					for (i=0;i<Nx;i++)
					{
						if (h_CRU2[pos(i,j,k)].Jrel>Spark_Threshold && spark_clock[pos(i,j,k)]>Sparks_Interval)
						{
							num_spark = num_spark + 1.0;
							spark_clock[pos(i,j,k)] = 0.0;
						}
						spark_clock[pos(i,j,k)] += out_step*DT;
					}
				}
			}
			spark_rate = num_spark*200.0/1.8/((Nx-2)*(Ny-2)*(Nz-2))/(out_step*DT/1000.0);

			////////////////////////////// output to screen /////////////////////////////
			end_time=clock()/(1.0*CLOCKS_PER_SEC);	
			printf(	"t=%g\t/ %g\t\ttime = %.1fs = %.1fh\t\tcit = %g\t\tcjsrt = %g\n",
					step*DT, stoptime, 
					end_time-start_time, (end_time-start_time)/3600.0, 
					cit, cjsrt
				  );

			////////////////////////////// whole cell ////////////////////////////////// flag
			fprintf(wholecell_file,	"%g %g %g %g %g " "%g %g %g %g %g "
									"%g %g %g %g %g " "%g %g %g %g %g "
									"%g %g %g %g %g " "%g %g %g %g %g "
									"%g %g %g %g %g " "%g\n",

									step*DT, cit,
									cpt, cst, 
									cjsrt, cnsrt, 
									v, I_NCX,
									I_Ca, Juptaket, 

									nai, I_Ks,
									I_Kr, I_K1,
									I_NaK, I_tos,
									I_tof, I_Na,
									Jleakt, Jrelt, 

									nout/(1.0*Nxyz), nobt/(1.0*Nxyz),
									ncut/(1.0*Nxyz), ncbt/(1.0*Nxyz),
									ncxflux*(Vs/Vp), icaflux, 
									I_bg, Kat, 
									TotalCa - TotalCa_before, CaExt, 

									nlcc_open, nlcc_flip,
									nlcc_act, nlcc_v,
									nlcc_Ca, spark_rate
					);

			fflush( wholecell_file );
			TotalCa_before = TotalCa;

			////////////////////////////// Line Scan ////////////////////////////////////	flag
			#ifdef output_linescan
				// if ( step*DT > ( stopbeat - 2 )*PCL && step*DT < ( stopbeat + 2 )*PCL )
				if (1)
				{
					for (i =1; i < Nx-1; i++)
					{
						int k = 4, j = Ny/2;
						ps = pos(i,j,k);
						fprintf(linescan_file, 	"%g %g %g %g %g " "%g %g %g %g %g "
												"%g %g %i %i %i " "%i %i %g \n",
												
												step*DT,			(double)i,
												h_CYT[ps*Nci].ci, 	h_CRU2[ps].cp,
												h_CRU[ps].cs, 		h_CRU2[ps].cjsr,
												h_CYT[ps*Nci].cnsr,	h_CRU2[ps].Jrel,
												h_CYT[ps*Nci].Juptake,	h_CRU2[ps].Jleak,

												h_CRU[ps].JCa, 		h_CRU[ps].JNCX,
												h_CRU2[ps].nou, 	h_CRU2[ps].nob, 
												h_CRU2[ps].ncu, 	h_CRU2[ps].ncb, 
												h_CRU2[ps].nLCC_open,  h_CRU[ps].Jbg
											 
							);
					}
					fprintf(linescan_file, "\n");
					fflush(linescan_file);
				}
			#endif
			
		}

		Compute<<<numBlocks, threadsPerBlock>>>( d_CRU, d_CRU2, d_CYT, d_CBU, d_SBU, v, step, nai);
		Finish<<<numBlocks, threadsPerBlock>>>( d_CRU, d_CRU2, d_CYT, d_CBU, d_SBU);

		step++;
	}

	fclose(wholecell_file);
	
	#ifdef output_linescan
		fclose(linescan_file);
	#endif


	hipFree(d_CYT);
	hipFree(d_CRU);
	hipFree(d_CRU2);
	hipFree(d_SBU);
	hipFree(d_CBU);
	
	free(h_CYT);
	free(h_CRU);
	free(h_CRU2);
	free(h_SBU);
	free(h_CBU);
	free(spark_clock);
	
	return EXIT_SUCCESS;
}

__global__ void Initial( cru *CRU, cru2 *CRU2, cytosol *CYT, cyt_bu *CBU, sl_bu *SBU, double ci_b, double cj_b)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int ps = pos(i,j,k);
	
	for ( int ix = 0; ix < Nci; ++ix )
	{
		int psi = ps*Nci+ix;
		CYT[psi].ci = ci_b;
		CYT[psi].cnsr = cj_b;
		CYT[psi].cinext = ci_b;
		CYT[psi].cnsrnext = cj_b;

		CBU[psi].catf = ktfon*ci_b*Btf/(ktfon*ci_b+ktfoff);
		CBU[psi].cats = ktson*ci_b*Bts/(ktson*ci_b+ktsoff);
		CBU[psi].cacal= kcalon*ci_b*Bcal/(kcalon*ci_b+kcaloff);
		CBU[psi].casr = ksron*ci_b*Bsr/(ksron*ci_b+ksroff);
		CBU[psi].cadye = kdyeon*ci_b*Bdye/(kdyeon*ci_b+kdyeoff);
		CBU[psi].cadyenext = kdyeon*ci_b*Bdye/(kdyeon*ci_b+kdyeoff);
	
		double ratio = Mgi*Kmyoca/(ci_b*Kmyomg);
		CBU[psi].camyo = ci_b*Bmyo/(Kmyoca+ci_b*(ratio+1.0));
		CBU[psi].mgmyo = CBU[psi].camyo*ratio;
	}
	
	SBU[ps].casar = ksaron*ci_b*Bsar/(ksaron*ci_b+ksaroff);
	SBU[ps].casarh = ksarhon*ci_b*Bsarh/(ksarhon*ci_b+ksarhoff);
	SBU[ps].cadye = kdyeon*ci_b*Bdye/(kdyeon*ci_b+kdyeoff);
	SBU[ps].cadyenext = kdyeon*ci_b*Bdye/(kdyeon*ci_b+kdyeoff);

	SBU[ps].casarj = ksaron*ci_b*Bsar/(ksaron*ci_b+ksaroff);
	SBU[ps].casarhj= ksarhon*ci_b*Bsarh/(ksarhon*ci_b+ksarhoff);
	SBU[ps].cadyej = kdyeon*ci_b*Bdye/(kdyeon*ci_b+kdyeoff);
	SBU[ps].cadyejnext = kdyeon*ci_b*Bdye/(kdyeon*ci_b+kdyeoff);

	CRU[ps].cs = ci_b;
	CRU[ps].csnext = ci_b;
	CRU2[ps].cp = ci_b;
	CRU2[ps].cpnext = ci_b;
	CRU2[ps].cjsr = cj_b;
	CRU2[ps].Tcj = cj_b + BCSQN*nCa*cj_b/( Kc+cj_b );

	CRU[ps].Jbg= 0;
	CRU[ps].JCa = 0;
	CRU[ps].JNCX = 0;
	CRU2[ps].Jrel = 0;

	for(int ll=0; ll<NLCC; ll++)
	{
		CRU2[ps].lcc[ll]=3;
	}

	double cb=BCSQN*nCa*CRU2[ps].cjsr/(Kc+CRU2[ps].cjsr);
	double ku2b = 1.0/( 1.0+pow(cb/BCSQN/(nCa/2.33), 24) )/taub;
	double kb2u = 1.0/tauu;
	double fracbound = 1/(1+kb2u/ku2b);

	CRU2[ps].nLCC_open = 0;
	CRU2[ps].ncb = int(fracbound*nryr);
	CRU2[ps].ncu = nryr-int(fracbound*nryr);
	CRU2[ps].nob = 0;
	CRU2[ps].nou = 0;

	CRU2[ps].Ka = 0.025;
}


#define FINESTEP 5
#define DTF 	(DT/FINESTEP)

__global__ void Compute( cru *CRU, cru2 *CRU2, cytosol *CYT, cyt_bu *CBU, sl_bu *SBU, double v, int step, double nai)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int ps = pos(i,j,k);

	hiprandState localState;
	localState=CRU2[ps].state;

	if ((i*j*k)!=0 && i<Nx-1 && j<Ny-1 && k<Nz-1)
	{
		//////////////////////////////////////////////////////////////////////
		/////////////////////////////////// ICa //////////////////////////////
		//////////////////////////////////////////////////////////////////////
		#ifndef Permeabilized
		{
			CRU2[ps].nLCC_open = 0;
			for (int  LCC_ichannel=0; LCC_ichannel<NLCC; LCC_ichannel++ )
			{
				CRU2[ps].lcc[LCC_ichannel] = LCCgating(v, CRU2[ps].cp, &localState, CRU2[ps].lcc[LCC_ichannel]);
				if ( CRU2[ps].lcc[LCC_ichannel] == 0 )
				{
					CRU2[ps].nLCC_open++;
				}
			}

			double ica = Single_LCC_Current(v, CRU2[ps].cp/1000.0 );
			
			CRU[ps].JCa = CRU2[ps].nLCC_open * ica;
		}
		#endif

		//////////////////////////////////////////////////////////////////////
		/////////////////////////////// INCX ////////////////////////////////
		//////////////////////////////////////////////////////////////////////
		#ifndef Permeabilized
			CRU[ps].JNCX = ncx( v, CRU[ps].cs, nai, &CRU2[ps].Ka );
		#endif

		//////////////////////////////////////////////////////////////////////
		/////////////////////////////// RyR //////////////////////////////////
		//////////////////////////////////////////////////////////////////////
		int Nryr_Open = ryrgating( CRU2[ps].cp, CRU2[ps].cjsr, &localState, &CRU2[ps].ncu, &CRU2[ps].nou, 
								&CRU2[ps].ncb, &CRU2[ps].nob, i, j, k, step );
		CRU2[ps].Jrel = Nryr_Open * f_Jmax*0.000147 * (CRU2[ps].cjsr-CRU2[ps].cp)/Vp;
		

		CRU2[ps].Jleak = 0;// Vleak * (CRU2[ps].cjsr - CRU2[ps].cp); // Shannon et al 2004, Eq.107
		//////////////////////////////////////////////////////////////////////
		/////////////////////// other currents ////////////////////////////
		//////////////////////////////////////////////////////////////////////

		CRU[ps].Jbg = 0;//0.00009425*(v-log(1800/CRU[ps].cs)/2.0/FRT);

		//////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////		
		//////////////////////////////////////////////////////////////////////
		double diffjn0 = (CRU2[ps].cjsr-CYT[ps*Nci].cnsr)/(tautr*2.0);
		double diffjn1 = (CRU2[ps].cjsr-CYT[ps*Nci+4].cnsr)/(tautr*2.0);
		
		double diffpi0 = (CRU2[ps].cp-CYT[ps*Nci].ci)/(taupi*2.0);
		double diffpi1 = (CRU2[ps].cp-CYT[ps*Nci+4].ci)/(taupi*2.0);
		double diffsi0 = (CRU[ps].cs-CYT[ps*Nci].ci)/(tausi*2.0);
		double diffsi1 = (CRU[ps].cs-CYT[ps*Nci+4].ci)/(tausi*2.0);

		double diffpidye0 = (SBU[ps].cadyej-CBU[ps*Nci].cadye)/(Ddye*taupi*2.0);
		double diffpidye1 = (SBU[ps].cadyej-CBU[ps*Nci+4].cadye)/(Ddye*taupi*2.0);

		double diffsidye0 = (SBU[ps].cadye-CBU[ps*Nci].cadye)/(Ddye*tausi*2.0);
		double diffsidye1 = (SBU[ps].cadye-CBU[ps*Nci+4].cadye)/(Ddye*tausi*2.0);
		

		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////// dotci ////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for ( int kz=0; kz<Niz; kz++ )
		{
			for ( int jy=0; jy<Niy; jy++ )
			{
				for ( int ix=0; ix<Nix; ix++ )
				{
					int psi = ps*Nci + posi(ix,jy,kz);
					int crui = posi(ix,jy,kz);

					CYT[psi].Juptake = uptake(CYT[psi].ci, CYT[psi].cnsr);
					CYT[psi].Jxileak = 0.00001035*2*(CYT[psi].cnsr-CYT[psi].ci)/(1.0+pow2(500.0/CYT[psi].cnsr));

					double bufftf = ktfon*CYT[psi].ci*(Btf-CBU[psi].catf) - ktfoff*CBU[psi].catf;
					double buffts = ktson*CYT[psi].ci*(Bts-CBU[psi].cats) - ktsoff*CBU[psi].cats;
					double buffcal = kcalon*CYT[psi].ci*(Bcal-CBU[psi].cacal) - kcaloff*CBU[psi].cacal;
					double buffsr = ksron*CYT[psi].ci*(Bsr-CBU[psi].casr) - ksroff*CBU[psi].casr;
					double buffmyo = konmyoca*CYT[psi].ci*(Bmyo-CBU[psi].camyo-CBU[psi].mgmyo)-koffmyoca*CBU[psi].camyo;
					double buffmyomg = konmyomg*Mgi*(Bmyo-CBU[psi].camyo-CBU[psi].mgmyo)-koffmyomg*CBU[psi].mgmyo;
					double buffdye = kdyeon*CYT[psi].ci*(Bdye-CBU[psi].cadye) - kdyeoff*CBU[psi].cadye;

					int inext =	  (ix==Nix-1)?( pos(i+1,j,k)*Nci+posi(0,jy,kz) ):( ps*Nci+posi(ix+1,jy,kz) );
					int ibefore = (ix==0)?( pos(i-1,j,k)*Nci+posi(Nix-1,jy,kz) ):( ps*Nci+posi(ix-1,jy,kz) );
					int jnext =	  (jy==Niy-1)?( pos(i,j+1,k)*Nci+posi(ix,0,kz) ):( ps*Nci+posi(ix,jy+1,kz) );
					int jbefore = (jy==0)?( pos(i,j-1,k)*Nci+posi(ix,Niy-1,kz) ):( ps*Nci+posi(ix,jy-1,kz) );
					int knext =   (kz==Niz-1)?( pos(i,j,k+1)*Nci+posi(ix,jy,0) ):( ps*Nci+posi(ix,jy,kz+1) );
					int kbefore = (kz==0)?( pos(i,j,k-1)*Nci+posi(ix,jy,Niz-1) ):( ps*Nci+posi(ix,jy,kz-1) );

					double coupleci =  	(CYT[knext].ci-CYT[psi].ci)/(tauit) +
										(CYT[kbefore].ci-CYT[psi].ci)/(tauit) +
										(CYT[jnext].ci-CYT[psi].ci)/(tauit) +
										(CYT[jbefore].ci-CYT[psi].ci)/(tauit) +
										(CYT[inext].ci-CYT[psi].ci)/(tauil) +
										(CYT[ibefore].ci-CYT[psi].ci)/(tauil);

					double couplecnsr = (CYT[knext].cnsr-CYT[psi].cnsr)/(taunt) +
										(CYT[kbefore].cnsr-CYT[psi].cnsr)/(taunt) +
										(CYT[jnext].cnsr-CYT[psi].cnsr)/(taunt) +
										(CYT[jbefore].cnsr-CYT[psi].cnsr)/(taunt) +
										(CYT[inext].cnsr-CYT[psi].cnsr)/(taunl) +
										(CYT[ibefore].cnsr-CYT[psi].cnsr)/(taunl);

					double coupledye = 	(CBU[knext].cadye-CBU[psi].cadye)/(Ddye*tauit) +
										(CBU[kbefore].cadye-CBU[psi].cadye)/(Ddye*tauit) +
										(CBU[jnext].cadye-CBU[psi].cadye)/(Ddye*tauit) +
										(CBU[jbefore].cadye-CBU[psi].cadye)/(Ddye*tauit) +
										(CBU[inext].cadye-CBU[psi].cadye)/(Ddye*tauil) +
										(CBU[ibefore].cadye-CBU[psi].cadye)/(Ddye*tauil);

					CYT[psi].cinext = CYT[psi].ci 
									  +(	- CYT[psi].Juptake + CYT[psi].Jxileak
											- bufftf - buffts - buffcal - buffsr - buffmyo - buffdye
											+ coupleci
											+( (crui==0)?( Vs/Vi*diffsi0+Vp/Vi*diffpi0 ):0 )
											+( (crui==4)?( Vs/Vi*diffsi1+Vp/Vi*diffpi1 ):0 ) 
									  )*DT;
					
					CYT[psi].cnsrnext = CYT[psi].cnsr 
										+(  CYT[psi].Juptake * Vi/Vnsr - CYT[psi].Jxileak * Vi/Vnsr
											+ couplecnsr
											+( (crui==0)?( diffjn0*Vjsr/Vnsr ):0 	)
											+( (crui==4)?( diffjn1*Vjsr/Vnsr ):0 	)
										)*DT;
					
					
					CBU[psi].catf += bufftf*DT;
					CBU[psi].cats += buffts*DT;
					CBU[psi].cacal += buffcal*DT;
					CBU[psi].casr += buffsr*DT;
					CBU[psi].camyo += buffmyo*DT;
					CBU[psi].mgmyo += buffmyomg*DT;
					CBU[psi].cadyenext += ( buffdye + coupledye
											+( (crui==0)?( (Vs/Vi)*(diffsidye0)+(Vp/Vi)*(diffpidye0) ):0 )
											+( (crui==4)?( (Vs/Vi)*(diffsidye1)+(Vp/Vi)*(diffpidye1) ):0 ) )*DT;

					if (CYT[psi].cinext < 0 ) 				CYT[psi].cinext = 1e-6;
					if (CYT[psi].cnsrnext < 0 )			CYT[psi].cnsrnext = 1e-6;
					if( CBU[psi].catf < 0 )				CBU[psi].catf = 1e-6;
					if( CBU[psi].cats < 0 )				CBU[psi].cats = 1e-6;
					if( CBU[psi].cacal < 0 )			CBU[psi].cacal = 1e-6;
					if( CBU[psi].casr < 0 )				CBU[psi].casr = 1e-6;
					if( CBU[psi].camyo < 0 )			CBU[psi].camyo = 1e-6;
					if( CBU[psi].mgmyo < 0 )			CBU[psi].mgmyo = 1e-6;
					if( CBU[psi].cadyenext < 0 )		CBU[psi].cadyenext = 1e-6;
				}
			}
		}
			


		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		for( int iii = 0; iii < FINESTEP; ++iii )
		{
			////////////////////// submembrane: dotcs ///////////////////////// 
			double csdiff = ( CRU[pos(i,j,k+1)].cs+CRU[pos(i,j,k-1)].cs-2*CRU[ps].cs )/(taust);
			double csdiffdye = ( SBU[pos(i,j,k+1)].cadye+SBU[pos(i,j,k-1)].cadye-2*SBU[ps].cadye )/(Ddye*taust);

			double diffps = (CRU2[ps].cpnext-CRU[ps].csnext)/taups;
			double diffpsdye = ( SBU[ps].cadyejnext - SBU[ps].cadyenext )/(taups*Ddye);

			double buffsar = ksaron*CRU[ps].csnext*(Bsar-SBU[ps].casar) - ksaroff*SBU[ps].casar;
			double buffsarh= ksarhon*CRU[ps].csnext*(Bsarh-SBU[ps].casarh) - ksarhoff*SBU[ps].casarh;
			double buffdye = kdyeon*CRU[ps].csnext*(Bdye-SBU[ps].cadyenext) - kdyeoff*SBU[ps].cadyenext;

			SBU[ps].casar += buffsar*DTF;
			SBU[ps].casarh += buffsarh*DTF;

			SBU[ps].cadyenext += DTF * ( buffdye + diffpsdye*Vp/Vs - diffsidye0 - diffsidye1 + csdiffdye );
			CRU[ps].csnext += DTF*( CRU[ps].JNCX - CRU[ps].Jbg * Vp/Vs
									+ diffps*Vp/Vs - diffsi0 - diffsi1 + csdiff
									- buffsar - buffsarh - buffdye );
			
			////////////////////// proximal space: dotcp ////////////////////// 
			buffsar = ksaron*CRU2[ps].cpnext*(Bsar-SBU[ps].casarj) - ksaroff*SBU[ps].casarj;
			buffsarh = ksarhon*CRU2[ps].cpnext*(Bsarh-SBU[ps].casarhj) - ksarhoff*SBU[ps].casarhj;
			buffdye = kdyeon*CRU2[ps].cpnext*(Bdye-SBU[ps].cadyejnext) - kdyeoff*SBU[ps].cadyejnext;

			SBU[ps].casarj += buffsar*DTF;
			SBU[ps].casarhj += buffsarh*DTF;

			SBU[ps].cadyejnext += DTF*( buffdye - diffpsdye - diffpidye0 - diffpidye1 );
			CRU2[ps].cpnext += DTF*( CRU2[ps].Jrel + CRU2[ps].Jleak - CRU[ps].JCa 
									- diffps - diffpi0 - diffpi1 
									- buffsar - buffsarh - buffdye );
		
			
			if ( CRU[ps].csnext < 0)			CRU[ps].csnext = 1e-6;
			if ( SBU[ps].casar < 0 )			SBU[ps].casar = 1e-6;
			if ( SBU[ps].casarh < 0 )			SBU[ps].casarh = 1e-6;
			if ( SBU[ps].cadyenext < 0 )		SBU[ps].cadyenext = 1e-6;

			if ( CRU2[ps].cpnext < 0 ) 			CRU2[ps].cpnext = 1e-6;
			if ( SBU[ps].casarj < 0 )			SBU[ps].casarj = 1e-6;
			if ( SBU[ps].casarhj < 0 )			SBU[ps].casarhj = 1e-6;
			if ( SBU[ps].cadyejnext < 0 )		SBU[ps].cadyejnext = 1e-6;

		}


		// dotcjsr
		double betaCSQN = 1.0/( 1.0 + BCSQN*Kc*nCa/pow2(Kc+CRU2[ps].cjsr) );
		CRU2[ps].cjsr += betaCSQN*( -diffjn0-diffjn1 - CRU2[ps].Jrel*Vp/Vjsr - CRU2[ps].Jleak*Vp/Vjsr )*DT;
		CRU2[ps].Tcj = CRU2[ps].cjsr + BCSQN*nCa*CRU2[ps].cjsr/(Kc+CRU2[ps].cjsr);

		CRU2[ps].state = localState;
	}
}


__global__ void Finish( cru *CRU, cru2 *CRU2, cytosol *CYT, cyt_bu *CBU, sl_bu *SBU )
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	int ps = pos(i,j,k);
	int ix, jy, kz, psb;

	if((i*j*k)!=0 && i<Nx-1 && j<Ny-1 && k<Nz-1)
	{
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////// update ////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		// this update should not be in the function Compute because of synchronization.
		CRU[ps].cs = CRU[ps].csnext;
		SBU[ps].cadye = SBU[ps].cadyenext;

		CRU2[ps].cp = CRU2[ps].cpnext;
		SBU[ps].cadyej=SBU[ps].cadyejnext;

		for( ix = 0; ix < Nci; ix++ )
		{
			CYT[ps*Nci+ix].ci = CYT[ps*Nci+ix].cinext;
			CYT[ps*Nci+ix].cnsr = CYT[ps*Nci+ix].cnsrnext;
			CBU[ps*Nci+ix].cadye = CBU[ps*Nci+ix].cadyenext;
			
		}
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		/////////////////////////////////////////////////////// Boundary //////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		#ifndef PermeabilizedB

			if (i==1)
			{
				psb=pos(0,j,k);
				for (jy=0;jy<Niy;jy++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(Nix-1,jy,kz)].cnsr = CYT[ps*Nci+posi(0,jy,kz)].cnsr;
						CYT[psb*Nci+posi(Nix-1,jy,kz)].ci  =  CYT[ps*Nci+posi(0,jy,kz)].ci;
						CBU[psb*Nci+posi(Nix-1,jy,kz)].cadye= CBU[ps*Nci+posi(0,jy,kz)].cadye;
					}
				}
			}

			if (i==Nx-2)
			{
				psb=pos(Nx-1,j,k);
				for (jy=0;jy<Niy;jy++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(0,jy,kz)].cnsr = CYT[ps*Nci+posi(Nix-1,jy,kz)].cnsr;
						CYT[psb*Nci+posi(0,jy,kz)].ci  =  CYT[ps*Nci+posi(Nix-1,jy,kz)].ci;
						CBU[psb*Nci+posi(0,jy,kz)].cadye= CBU[ps*Nci+posi(Nix-1,jy,kz)].cadye;
					}
				}
			}

			if (j==1)
			{
				psb=pos(i,0,k);
				for (ix=0;ix<Nix;ix++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(ix,Niy-1,kz)].cnsr = CYT[ps*Nci+posi(ix,0,kz)].cnsr;
						CYT[psb*Nci+posi(ix,Niy-1,kz)].ci  =  CYT[ps*Nci+posi(ix,0,kz)].ci;
						CBU[psb*Nci+posi(ix,Niy-1,kz)].cadye= CBU[ps*Nci+posi(ix,0,kz)].cadye;
					}
				}
			}

			if (j==Ny-2)
			{
				psb=pos(i,Ny-1,k);
				for (ix=0;ix<Nix;ix++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(ix,0,kz)].cnsr = CYT[ps*Nci+posi(ix,Niy-1,kz)].cnsr;
						CYT[psb*Nci+posi(ix,0,kz)].ci  =  CYT[ps*Nci+posi(ix,Niy-1,kz)].ci;
						CBU[psb*Nci+posi(ix,0,kz)].cadye= CBU[ps*Nci+posi(ix,Niy-1,kz)].cadye;
					}
				}
			}

			if (k==1)
			{
				psb=pos(i,j,0);
				for (ix=0;ix<Nix;ix++)
				{
					for (jy=0;jy<Niy;jy++)
					{
						CYT[psb*Nci+posi(ix,jy,Niz-1)].cnsr = CYT[ps*Nci+posi(ix,jy,0)].cnsr;
						CYT[psb*Nci+posi(ix,jy,Niz-1)].ci  =  CYT[ps*Nci+posi(ix,jy,0)].ci;
						CBU[psb*Nci+posi(ix,jy,Niz-1)].cadye= CBU[ps*Nci+posi(ix,jy,0)].cadye;
					}
				}
				CRU[psb].cs = CRU[ps].cs;
				SBU[psb].cadye = SBU[ps].cadye;
			}

			if (k==Nz-2)
			{
				psb=pos(i,j,Nz-1);
				for (ix=0;ix<Nix;ix++)
				{
					for (jy=0;jy<Niy;jy++)
					{
						CYT[psb*Nci+posi(ix,jy,0)].cnsr = CYT[ps*Nci+posi(ix,jy,Niz-1)].cnsr;
						CYT[psb*Nci+posi(ix,jy,0)].ci  =  CYT[ps*Nci+posi(ix,jy,Niz-1)].ci;
						CBU[psb*Nci+posi(ix,jy,0)].cadye= CBU[ps*Nci+posi(ix,jy,Niz-1)].cadye;
					}
				}
				CRU[psb].cs = CRU[ps].cs;
				SBU[psb].cadye = SBU[ps].cadye;
			}

		#else // Permeabilized cell
			if (i==1)
			{
				psb=pos(0,j,k);
				for (jy=0;jy<Niy;jy++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(Nix-1,jy,kz)].cnsr = CYT[ps*Nci+posi(0,jy,kz)].cnsr;
					}
				}
			}

			if (i==Nx-2)
			{
				psb=pos(Nx-1,j,k);
				for (jy=0;jy<Niy;jy++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(0,jy,kz)].cnsr = CYT[ps*Nci+posi(Nix-1,jy,kz)].cnsr;
					}
				}
			}

			if (j==1)
			{
				psb=pos(i,0,k);
				for (ix=0;ix<Nix;ix++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(ix,Niy-1,kz)].cnsr = CYT[ps*Nci+posi(ix,0,kz)].cnsr;
					}
				}
			}

			if (j==Ny-2)
			{
				psb=pos(i,Ny-1,k);
				for (ix=0;ix<Nix;ix++)
				{
					for (kz=0;kz<Niz;kz++)
					{
						CYT[psb*Nci+posi(ix,0,kz)].cnsr = CYT[ps*Nci+posi(ix,Niy-1,kz)].cnsr;
					}
				}
			}

			if (k==1)
			{
				psb=pos(i,j,0);
				for (ix=0;ix<Nix;ix++)
				{
					for (jy=0;jy<Niy;jy++)
					{
						CYT[psb*Nci+posi(ix,jy,Niz-1)].cnsr = CYT[ps*Nci+posi(ix,jy,0)].cnsr;
					}
				}
			}

			if (k==Nz-2)
			{
				psb=pos(i,j,Nz-1);
				for (ix=0;ix<Nix;ix++)
				{
					for (jy=0;jy<Niy;jy++)
					{
						CYT[psb*Nci+posi(ix,jy,0)].cnsr = CYT[ps*Nci+posi(ix,jy,Niz-1)].cnsr;
					}
				}
			}
		#endif
	}
}

__global__ void	setup_kernel(unsigned long long seed, cru2 *CRU2 )
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	hiprand_init(seed, pos(i,j,k), 0, &(CRU2[pos(i,j,k)].state)	);
}


__device__ int ryrgating(double cp, double cjsr, hiprandState *state, int *ncu, int *nou, int *ncb, int *nob, int i, int j, int k, int step)
{
	hiprandState localState = *state;

	// should be satisfied: ku*DT < 1.0
	double ku = 4.0 /( 1.0 + pow2(5000.0/cjsr) )/( 1.0 + pow2(3.125/cp) );
	double kb = 2.52/( 1.0 + pow2(5000.0/cjsr) )/( 1.0 + pow2(3.125/cp) );

	double kuminus = 1.0/taucu;
	double kbminus = 1.0/taucb;
	
	double cb = BCSQN * nCa * cjsr / (Kc+cjsr); // bound Ca2+ concentration in JSR
	double ku2b = 1.0/( 1.0+pow(cb/BCSQN/(nCa/2.33), 24) )/taub;
	double kb2u = 1.0/tauu;
	
	double p_cu_ou = ku * DT;
	double p_cb_ob = kb * DT;
	double p_ou_cu = kuminus * DT;
	double p_ob_cb = kbminus * DT;
	double p_ou_ob = ku2b * DT;
	double p_cu_cb = ku2b * DT;
	double p_cb_cu = kb2u * DT;
	double p_ob_ou = kb2u * (ku/kb) * DT;

	if ( kb < 1e-16 )
	{
		p_ou_ob = 0;
		p_ob_ou = 0;
	}

	int n_cu_ou = number_RyR_transit( &localState, *ncu, p_cu_ou, *ncu );
	int n_cu_cb = number_RyR_transit( &localState, *ncu, p_cu_cb, *ncu - n_cu_ou );
	int n_ou_cu = number_RyR_transit( &localState, *nou, p_ou_cu, *nou );
	int n_ou_ob = number_RyR_transit( &localState, *nou, p_ou_ob, *nou - n_ou_cu );
	int n_cb_cu = number_RyR_transit( &localState, *ncb, p_cb_cu, *ncb );
	int n_cb_ob = number_RyR_transit( &localState, *ncb, p_cb_ob, *ncb - n_cb_cu );
	int n_ob_ou = number_RyR_transit( &localState, *nob, p_ob_ou, *nob );
	int n_ob_cb = number_RyR_transit( &localState, *nob, p_ob_cb, *nob - n_ob_ou );

	*nou += - n_ou_ob - n_ou_cu + n_ob_ou + n_cu_ou;
	*nob += - n_ob_ou - n_ob_cb + n_ou_ob + n_cb_ob;
	*ncu += - n_cu_ou - n_cu_cb + n_ou_cu + n_cb_cu;
	*ncb += - n_cb_cu - n_cb_ob + n_cu_cb + n_ob_cb;

	*state = localState;

	return ( *nou + *nob );
}


// NN: number of RyRs in the current state
// probability: the probability to transit to another state
// upBound: maximum number of RyRs to transit
__device__ int number_RyR_transit(hiprandState *state, int NN, double probability, int upBound)
{
	int Ntransit = 1001; // larger than nryr
	double mean = NN*probability;

	// If the condition is satisfied, Ntransit is a poisson distribution,
	// otherwise it is a gaussian distribution. They are the approximations 
	// of the binomial distribution.
	if ( probability < 0.26*exp(-NN/2.245) + 0.12*exp(-NN/35.17) + 0.11 )
		while ( Ntransit > upBound ) // Poisson random nmber
		{
			int k = 0;
			double p = 1.0;
			while ( p >= exp(-mean) )
			{
				k++;
				p = p * hiprand_uniform_double(state);
			}
			Ntransit = k - 1;
		}
	else
		while ( Ntransit < 0 || Ntransit > upBound )
			Ntransit = lrintf( mean + sqrt( mean * (1.0-probability) ) * hiprand_normal_double(state) );

	return Ntransit;
}


__device__ int LCCgating(double v, double cp, hiprandState *state, int i )
{	

	hiprandState localState=*state;

	double dv5 = 5;
	double dvk = 8;

	double fv5 = -22.8;	
	double fvk = 9.1;

	double alphac = 0.22;
	double betac = 4;

	#ifdef ISO
		betac = 2;
		dv5 = 0;
		fv5 = -28;
		fvk = 8.5;
	#endif

	double dinf = 1.0/(1.0+exp(-(v-dv5)/dvk));
	double taud_inverse = 1.0/((1.0-exp(-(v-dv5)/dvk))/(0.035*(v-dv5))*dinf);
	if( (v > -0.0001) && (v < 0.0001) )
		taud_inverse = 0.035*dvk/dinf;
	
	double finf = 1.0-1.0/(1.0+exp(-(v-fv5)/fvk))/(1.+exp((v-60)/12.));
	double tauf_inverse = (0.02-0.007*exp(-pow2(0.0337*(v+10.5))));
	

	double alphad = dinf * taud_inverse;
	double betad = (1.0-dinf) * taud_inverse;
	
	double alphaf = finf * tauf_inverse;
	double betaf = (1.0-finf) * tauf_inverse;
	
	double alphafca = 0.006;
	double betafca = 0.175/( 1 + pow2(35.0/cp) );

	double random = hiprand_uniform_double(&localState)/DT;
	*state=localState;
	

	if ( i%2 )
		if ( random < alphac )
			return i-1;
		else
			random -= alphac;
	else
		if ( random < betac )
			return i+1;
		else
			random -= betac;
	

	if ( (i/2)%2 )
		if ( random < alphad )
			return i-2;
		else
			random -= alphad;
	else
		if ( random < betad )
			return i+2;
		else
			random -= betad;
	
	
	if ( (i/4)%2 )
		if ( random < alphaf )
			return i-4;
		else
			random -= alphaf;
	else
		if ( random < betaf )
			return i+4;
		else
			random -= betaf;
	
	
	if ( (i/8)%2 )
		if ( random < alphafca )
			return i-8;
		else
			random -= alphafca;
	else
		if ( random < betafca )
			return i+8;
		else
			random -= betafca;

	return (i);
}

__device__ double Single_LCC_Current(double v, double cp) // cp in mM
{
	double ica = 0;
	double za = v*Faraday/RR/Temperature;
	if ( fabs(za)<0.001 ) 
		ica = 2.0*Pca*Faraday*gammai*(cp*exp(2.0*za)-CaO);
	else 
		ica = 4.0*Pca*za*Faraday*gammai*(cp*exp(2.0*za)-CaO)/(exp(2.0*za)-1.0);

	if (ica > 0.0)
		ica = 0.0;

	return ( ica );
}

double Ina( double v, double *hh, double *jj, double *mm, double nai )
{
	double Ena = 1.0/FRT*log(NaO/nai);
	double am = 0.32*(v+47.13)/(1.0-exp(-0.1*(v+47.13)));
	double bm = 0.08*exp(-v/11.0);

	double ah,bh,aj,bj;

	if(v < -40.0)
	{
		ah = 0.135 * exp( -(80.0+v)/6.8 );
		bh = 3.56 * exp(0.079*v) + 310000.0*exp(0.35*v);
		aj = (-127140.0*exp(0.2444*v)-0.00003474*exp(-0.04391*v)) * ( (v+37.78)/(1.0+exp(0.311*(v+79.23))) );
		bj = (0.1212*exp(-0.01052*v))/(1.0+exp(-0.1378*(v+40.14)));
		
	}
	else
	{
		ah = 0.0;
		bh = 1.0/( 0.130*(1.0+exp((v+10.66)/(-11.1))) );
		aj = 0.0;
		bj = ( 0.3*exp(-0.0000002535*v) )/( 1.0 + exp(-0.1*(v+32.0)) );
				
	}
			
	double tauh = 1.0/(ah+bh);
	double tauj = 1.0/(aj+bj);
	double taum = 1.0/(am+bm);

	*hh = ah/(ah+bh)-((ah/(ah+bh))-*hh)*exp(-DT/tauh);
	*jj = aj/(aj+bj)-((aj/(aj+bj))-*jj)*exp(-DT/tauj);
	*mm = am/(am+bm)-((am/(am+bm))-*mm)*exp(-DT/taum);

	double INa = gNa*(alphaNaL+(1.0-alphaNaL)*(*hh))*(alphaNaL+(1.0-alphaNaL)*(*jj))*(*mm)*(*mm)*(*mm)*(v-Ena) + gNaLeak*(v-Ena);

	return INa;
}


double Ikr( double v, double *Xkr )
{
	double krv1 = 0.00138*(v+7.0)/( 1.0-exp(-0.123*(v+7.0))  );
	double krv2 = 0.00061*(v+10.0)/(exp( 0.145*(v+10.0))-1.0);
	double taukr = 1.0/(krv1+krv2);
	double Xkr_inf= 1.0/(1.0+exp(-(v+50.0)/7.5));
	double Rkr = 1.0/(1.0+exp((v+33.0)/22.4));

	*Xkr = Xkr_inf - ( Xkr_inf - *Xkr ) * exp(-DT/taukr);
	
	double I_Kr = gKr * sqrt(KO/5.40) * (*Xkr) * Rkr * (v-Ek);

	return I_Kr;
}

double Iks( double v, double *Xs1, double *Xs2, double *Qks, double cst, double nai )
{
	double prnak = 0.01833;
	double Eks = (1.0/FRT)*log((KO+prnak*NaO)/(KI+prnak*nai));

	double qks_inf = 0.2*( 1.0 + 0.8/( 1.0 + pow((0.28/cst),3) ) );
	double tauqks = 1000.0;

	double Xs1_inf= 1.0/(1.0+exp(-(v-1.5)/16.7));
	double tauxs = 1.2/( 0.0000719*(v+30.0)/(1.0-exp(-0.148*(v+30.0)))
							+ 0.000131*(v+30.0)/(exp(0.06870*(v+30.0))-1.0) );

	*Xs1 = Xs1_inf-(Xs1_inf-*Xs1)*exp(-DT/tauxs);
	*Xs2 = Xs1_inf-(Xs1_inf-*Xs2)*exp(-DT/tauxs);
	*Qks = *Qks + DT*( qks_inf-*Qks )/tauqks;

	double I_Ks = gKs*(*Qks)*(*Xs1)*(*Xs2)*(v-Eks);

	return I_Ks;
}

double Ik1( double v )
{
	double Aki = 1.02/(1.0+exp(0.2385*(v-Ek-59.215)));
	double Bki = (0.49124*exp(0.08032*(v-Ek+5.476))+exp(0.061750*(v-Ek-594.31)))/(1.0+exp(-0.5143*(v-Ek+4.753)));
	double I_K1 = gK1 * sqrt(KO/5.4) * Aki/(Aki+Bki) * (v-Ek);

	return I_K1;
}

double Itos(double v, double *Xtos, double *Ytos)
{
	double Xtos_inf = 1.0/( 1.0 + exp( -(v+3.0)/15.0) );
	double Ytos_inf = 1.0/( 1.0 + exp( (v+33.5)/10.0) );
	double Rs_inf = 1.0/( 1.0 + exp( (v+33.5)/10.0) );
	double txs = 9.0/( 1.0 + exp( (v+3.0)/15.0) ) + 0.5;
	double tys = 3000.0/(1.0+exp( (v+60.0)/10.0) ) + 30.0;

	*Xtos = Xtos_inf-(Xtos_inf-*Xtos)*exp(-DT/txs);
	*Ytos = Ytos_inf-(Ytos_inf-*Ytos)*exp(-DT/tys);

	double I_tos = gtos*(*Xtos)*(*Ytos+0.5*Rs_inf)*(v-Ek);

	return I_tos;
}

///////////////// Ito /////////////////

double Itof(double v, double *Xtof, double *Ytof)
{
	double Xtof_inf = 1.0/(1.0+exp( -(v+3.0)/15.0) );
	double Ytof_inf = 1.0/(1.0+exp( (v+33.5)/10.0) );
	double txf = 3.5 * exp( -(v/30.00)*(v/30.0) ) + 1.5;
	double tyf = 20.0/( 1.0+exp( (v+33.5)/10.0 ) )+20.0;

	*Xtof = Xtof_inf-(Xtof_inf-*Xtof)*exp(-DT/txf);
	*Ytof = Ytof_inf-(Ytof_inf-*Ytof)*exp(-DT/tyf);

	double I_tof = gtof*(*Xtof)*(*Ytof)*(v-Ek);

	return I_tof;
}


double Inak( double v, double nai )	 // Mahajan et al 2008
{
	double sigma = ( exp(NaO/67.3) - 1.0 )/7.0;
	double fNaK = 1.0/( 1.0 + 0.1245*exp(-0.1*v*FRT) + 0.0365*sigma*exp(-v*FRT) );
	double I_NaK = gNaK * fNaK * 1.0/( 1.0+pow(12.0/nai,1.0) ) * KO/(KO+1.5);

	return I_NaK;
}	

///////////////////////////	sodium dynamics /////////////////////////////////
double sodium(double v, double nai, double I_Na, double I_NaK, double I_NCX)
{
	// convert pA/pF to mM/ms. Mahajan et al 2008, Eq. 33
	double alpha = 1.0/(2.0*0.096485)/( Vi*Nci*(Nx-2)*(Ny-2)*(Nz-2) ) * Cm / 1000.0;
	double trick = 0.768; // just to speed up Nai dynamics

	double dnai = - trick * alpha * ( I_Na + 3.0*I_NaK + 3.0*I_NCX );

	return (nai + dnai*DT);
}

__device__ double ncx(double v, double cs, double nai, double *Ka)
{
	cs = cs/1000.0;
	double za = v*Faraday/RR/Temperature;

	double t1 = Kmcai*pow3(NaO)*( 1.0+pow3(nai/Kmnai) );
	double t2 = pow3(Kmnao)*cs*(1.0+cs/Kmcai);
	double t3 = (Kmcao+CaO)*pow3(nai) + cs*pow3(NaO);

	double dotKa = ( 1.0/(1.0+pow3(0.0003/cs)) - (*Ka) )/150.0;
	*Ka += dotKa*DT;
	// *Ka = 1.0/(1.0+pow3(0.0003/cs));

	double Inaca = Vncx * (*Ka) * ( exp(eta*za)*pow3(nai)*CaO-exp((eta-1.0)*za)*pow3(NaO)*cs )
					/((t1+t2+t3)*(1.0+ksat*exp((eta-1.0)*za)));


	return Inaca;
}	

__device__ double uptake(double ci, double cnsr)		//uptake
{
	double Ki = 0.123;
	double Knsr = 1700.0;
	double HH = 1.787;
	double Iuptake = Vup * (pow(ci/Ki,HH)-pow(cnsr/Knsr,HH)) / (1.0+pow(ci/Ki,HH)+pow(cnsr/Knsr,HH));
	return Iuptake;
}


void matrix2file(cytosol *CYT, int step)
{
	int i,j,k,ix,jy,kz; // i,j,k for CRU index; ix, jy, kz for lattices in each CRU
	double average=0;
	char FileName[50];
	sprintf(FileName,"%-s%d%s","step",step,".vtk");
	
	FILE * file_pointer;
	file_pointer=fopen(FileName,"w");
		fprintf(file_pointer, "# vtk DataFile Version 3.0\n");
		fprintf(file_pointer, "3d\n");
		fprintf(file_pointer, "ASCII\n");
		fprintf(file_pointer, "DATASET STRUCTURED_POINTS\n");
		fprintf(file_pointer, "DIMENSIONS %d %d %d\n",(Nx-2)*Nix,(Ny-2)*Niy,(Nz-2)*Niz);
		fprintf(file_pointer, "ASPECT_RATIO 1 1 1\n");
		fprintf(file_pointer, "ORIGIN 0 0 0\n");
		fprintf(file_pointer, "POINT_DATA %d\n",(Nx-2)*Nix*(Ny-2)*Niy*(Nz-2)*Niz);
		fprintf(file_pointer, "SCALARS ci double 1\n");
		fprintf(file_pointer, "LOOKUP_TABLE default\n\n");
		for (k=1;k<(Nz-1);k++)
		{
			for (kz=0;kz<Niz;kz++)
			{
				for (j=1;j<(Ny-1);j++)
				{
					for (jy=0;jy<Niy;jy++)
					{
						for (i=1;i<(Nx-1);i++)
						{
							for (ix=0;ix<Nix;ix++)
							{
								fprintf(file_pointer,"%g \t", CYT[pos(i,j,k)*Nci+posi(ix,jy,kz)].ci);
								average += CYT[pos(i,j,k)*Nci+posi(ix,jy,kz)].ci;
								if( CYT[pos(i,j,k)*Nci+posi(ix,jy,kz)].ci > 50.0 )
								{
									cout << step*DT <<" "<<i<<" "<<j<<" "<<k<<" "<<posi(ix,jy,kz)<<
									" error! ci="<< CYT[pos(i,j,k)*Nci+posi(ix,jy,kz)].ci << endl;
								}
							}
						}
						fprintf(file_pointer, "\n");
					}
				}
			}
		}
		average /= (1.0*(Nx-2)*(Ny-2)*(Nz-2)*Nix*Niy*Niz);
		printf("Average=%g\t",average);
	fclose(file_pointer);
}
